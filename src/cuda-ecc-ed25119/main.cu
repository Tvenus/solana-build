#include <stdio.h>
#include "ed25519.h"
#include <inttypes.h>
#include <assert.h>

#define LOG(...) if (verbose) { printf(__VA_ARGS__); }

bool verbose = false;

void print_dwords(unsigned char* ptr, int size) {
    for (int j = 0; j < (size)/(int)sizeof(uint32_t); j++) {
        LOG("%x ", ((uint32_t*)ptr)[j]);
    }
}

int main(int argc, const char* argv[]) {
    if (argc != 2 && argc != 3) {
        printf("usage: %s [-v] <num_signatures>\n", argv[0]);
        return 1;
    }
    const char* sig_ptr = argv[1];
    if (argc == 3) {
        if (0 == strcmp(argv[1], "-v")) {
            verbose = true;
            sig_ptr = argv[2];
        } else {
            printf("what is this? %s\n", argv[1]);
            return 1;
        }
    }
    int num_signatures = strtol(sig_ptr, NULL, 10);
    // Host allocate
    unsigned char* seed_h = (unsigned char*)calloc(num_signatures * SEED_SIZE, sizeof(uint32_t));
    unsigned char* public_key_h = (unsigned char*)calloc(num_signatures, PUB_KEY_SIZE);
    unsigned char* private_key_h = (unsigned char*)calloc(num_signatures, PRIV_KEY_SIZE);
    unsigned char* signature_h = (unsigned char*)calloc(num_signatures, SIG_SIZE);
    unsigned char message_h[] = "abcd1234";
    int message_h_len = strlen((char*)message_h);
    unsigned char* messages_h = (unsigned char*)calloc(num_signatures, message_h_len);
    uint32_t* message_lens_h = (uint32_t*)calloc(num_signatures, sizeof(uint32_t));
    uint32_t* message_offsets_h = (uint32_t*)calloc(num_signatures, sizeof(uint32_t));

    LOG("initing signatures..\n");
    for (int i = 0; i < num_signatures; i++) {
        message_lens_h[i] = message_h_len;
        memcpy(&messages_h[i * message_h_len], message_h, message_h_len);
        message_offsets_h[i] = i * message_h_len;
        LOG("message_len: %d offsets: %d message: %d\n",
               message_lens_h[i], message_offsets_h[i], messages_h[i * message_h_len]);
    }

    LOG("device allocate..\n");
    // Device allocate
    unsigned char* signature = NULL;
    hipMalloc(&signature, num_signatures * SIG_SIZE);

    unsigned char* messages = NULL;
    hipMalloc(&messages, num_signatures * message_h_len);

    LOG("device allocate2..\n");
    uint32_t* message_lens = NULL;
    hipMalloc(&message_lens, num_signatures * sizeof(uint32_t));

    uint32_t* message_offsets = NULL;
    hipMalloc(&message_offsets, num_signatures * sizeof(uint32_t));

    unsigned char* public_key = NULL;
    hipMalloc(&public_key, num_signatures * PUB_KEY_SIZE);

    unsigned char* private_key = NULL;
    hipMalloc(&private_key, num_signatures * PRIV_KEY_SIZE);

    int* out = NULL;
    int out_size = num_signatures * sizeof(int);
    hipMalloc(&out, out_size);

    int* out_h = (int*)calloc(1, out_size);

    LOG("creating seed..\n");
    int ret = ed25519_create_seed(seed_h);
    LOG("create_seed: %d\n", ret);
    ed25519_create_keypair(public_key_h, private_key_h, seed_h);
    ed25519_sign(signature_h, message_h, message_h_len, public_key_h, private_key_h);
    ret = ed25519_verify(signature_h, message_h, message_h_len, public_key_h);
    LOG("verify: %d\n", ret);

    for (int i = 1; i < num_signatures; i++) {
        memcpy(&signature_h[i * SIG_SIZE], signature_h, SIG_SIZE);
        memcpy(&public_key_h[i * PUB_KEY_SIZE], public_key_h, PUB_KEY_SIZE);
    }

    for (int i = 0; i < num_signatures; i++ ) {
        unsigned char* sig_ptr = &signature_h[i * SIG_SIZE];
        unsigned char* messages_ptr = &messages_h[i * message_h_len];
        print_dwords(sig_ptr, SIG_SIZE);
        LOG("\n");
        print_dwords(messages_ptr, message_h_len);
        LOG("\n");
    }
    LOG("\n");

    hipMemcpy(signature, signature_h, num_signatures * SIG_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(messages, messages_h, num_signatures * message_h_len, hipMemcpyHostToDevice);
    hipMemcpy(message_lens, message_lens_h, num_signatures * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(message_offsets, message_offsets_h, num_signatures * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(public_key, public_key_h, num_signatures * PUB_KEY_SIZE, hipMemcpyHostToDevice);

    ed25519_verify_many(signature, messages, message_lens, message_offsets, public_key, num_signatures, out);

    hipMemcpy(out_h, out, out_size, hipMemcpyDeviceToHost);
    LOG("ret:\n");
    bool verify_failed = false;
    for (int i = 0; i < out_size / (int)sizeof(int); i++) {
        LOG("%x ", out_h[i]);
        if (out_h[i] != 1) {
            verify_failed = true;
        }
    }
    LOG("\n");
    fflush(stdout);
    assert(verify_failed == false);
    return 0;
}
